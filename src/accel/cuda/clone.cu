#include "clone.h"


/**
 * @brief Given a pointer to a Material on the host and a dev_material on the
 *        GPU, copy all of the properties from the Material object on the host
 *        struct to the GPU.
 * @details This routine is called by the GPUSolver::initializeMaterials()
 *          private class method and is not intended to be called directly.
 * @param material_h pointer to a Material on the host
 * @param material_d pointer to a dev_material on the GPU
 */
void clone_material(Material* material_h, dev_material* material_d) {

  /* Copy over the Material's ID */
  int id = material_h->getId();
  int num_groups = material_h->getNumEnergyGroups();

  hipMemcpy((void*)&material_d->_id, (void*)&id, sizeof(int),
             hipMemcpyHostToDevice);

  /* Allocate memory on the device for each dev_material data array */
  FP_PRECISION* sigma_t;
  FP_PRECISION* sigma_s;
  FP_PRECISION* sigma_f;
  FP_PRECISION* nu_sigma_f;
  FP_PRECISION* chi;
  FP_PRECISION* fiss_matrix;

  /* Allocate memory on device for dev_material data arrays */
  hipMalloc((void**)&sigma_t, num_groups * sizeof(FP_PRECISION));
  hipMalloc((void**)&sigma_s, num_groups * num_groups * sizeof(FP_PRECISION));
  hipMalloc((void**)&sigma_f, num_groups * sizeof(FP_PRECISION));
  hipMalloc((void**)&nu_sigma_f, num_groups * sizeof(FP_PRECISION));
  hipMalloc((void**)&chi, num_groups * sizeof(FP_PRECISION));
  hipMalloc((void**)&fiss_matrix, num_groups * num_groups * sizeof(FP_PRECISION));

  /* Copy Material data from host to arrays on the device */
  hipMemcpy((void*)sigma_t, (void*)material_h->getSigmaT(),
             num_groups * sizeof(FP_PRECISION), hipMemcpyHostToDevice);
  hipMemcpy((void*)sigma_s, (void*)material_h->getSigmaS(),
             num_groups * num_groups * sizeof(FP_PRECISION),
             hipMemcpyHostToDevice);
  hipMemcpy((void*)sigma_f, (void*)material_h->getSigmaF(),
             num_groups * sizeof(FP_PRECISION), hipMemcpyHostToDevice);
  hipMemcpy((void*)nu_sigma_f, (void*)material_h->getNuSigmaF(),
             num_groups * sizeof(FP_PRECISION), hipMemcpyHostToDevice);
  hipMemcpy((void*)chi, (void*)material_h->getChi(),
             num_groups * sizeof(FP_PRECISION), hipMemcpyHostToDevice);
  hipMemcpy((void*)fiss_matrix, (void*)material_h->getFissionMatrix(),
             num_groups * num_groups * sizeof(FP_PRECISION),
             hipMemcpyHostToDevice);

  /* Copy Material data pointers to dev_material on GPU */
  hipMemcpy((void*)&material_d->_sigma_t, (void*)&sigma_t,
             sizeof(FP_PRECISION*), hipMemcpyHostToDevice);
  hipMemcpy((void*)&material_d->_sigma_s, (void*)&sigma_s,
             sizeof(FP_PRECISION*), hipMemcpyHostToDevice);
  hipMemcpy((void*)&material_d->_sigma_f, (void*)&sigma_f,
             sizeof(FP_PRECISION*), hipMemcpyHostToDevice);
  hipMemcpy((void*)&material_d->_nu_sigma_f, (void*)&nu_sigma_f,
             sizeof(FP_PRECISION*), hipMemcpyHostToDevice);
  hipMemcpy((void*)&material_d->_chi, (void*)&chi,
             sizeof(FP_PRECISION*), hipMemcpyHostToDevice);
  hipMemcpy((void*)&material_d->_fiss_matrix, (void*)&fiss_matrix,
             sizeof(FP_PRECISION*), hipMemcpyHostToDevice);

  return;
}


/**
 * @brief Given a pointer to a Track on the host, a dev_track on
 *        the GPU, and the map of material IDs to indices in the
 *        _materials array, copy all of the class attributes and
 *        segments from the Track object on the host to the GPU.
 * @details This routine is called by the GPUSolver::initializeTracks()
 *          private class method and is not intended to be called
 *          directly.
 * @param track_h pointer to a Track on the host
 * @param track_d pointer to a dev_track on the GPU
 * @param material_IDs_to_indices map of material IDs to indices
 *        in the _materials array.
 */
void clone_track(Track* track_h, dev_track* track_d,
     		 std::map<int, int> &material_IDs_to_indices) {

  dev_segment* dev_segments;
  dev_segment* host_segments = new dev_segment[track_h->getNumSegments()];
  dev_track new_track;

  new_track._uid = track_h->getUid();
  new_track._num_segments = track_h->getNumSegments();
  new_track._azim_angle_index = track_h->getAzimAngleIndex();
  new_track._next_in = track_h->isNextIn();
  new_track._next_out = track_h->isNextOut();
  new_track._transfer_flux_in = track_h->getTransferFluxIn();
  new_track._transfer_flux_out = track_h->getTransferFluxOut();

  hipMalloc((void**)&dev_segments,
             track_h->getNumSegments() * sizeof(dev_segment));
  new_track._segments = dev_segments;

  for (int s=0; s < track_h->getNumSegments(); s++) {
    segment* curr = track_h->getSegment(s);
    host_segments[s]._length = curr->_length;
    host_segments[s]._region_uid = curr->_region_id;
    host_segments[s]._material_index =
      material_IDs_to_indices[curr->_material->getId()];
  }

  hipMemcpy((void*)dev_segments, (void*)host_segments,
             track_h->getNumSegments() * sizeof(dev_segment),
             hipMemcpyHostToDevice);
  hipMemcpy((void*)track_d, (void*)&new_track, sizeof(dev_track),
             hipMemcpyHostToDevice);

  delete [] host_segments;

  return;
}
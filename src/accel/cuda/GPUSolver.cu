#include "hip/hip_runtime.h"
#include "GPUSolver.h"

/** The number of azimuthal angles */
__constant__ int num_azim[1];

/** The number of energy groups */
__constant__ int num_groups[1];

/** The number of FSRs */
__constant__ int num_FSRs[1];

/** The number of polar angles */
__constant__ int num_polar[1];

/** Half the number of polar angles */
__constant__ int num_polar_2[1];

/** The number of polar angles times energy groups */
__constant__ int polar_times_groups[1];

/** An array for the sines of the polar angle in the Quadrature set */
__constant__ FP_PRECISION sin_thetas[MAX_POLAR_ANGLES_GPU];

/** An array of the weights from the Quadrature set */
__constant__ FP_PRECISION weights[MAX_POLAR_ANGLES_GPU*MAX_AZIM_ANGLES_GPU];

/** The total number of Tracks */
__constant__ int tot_num_tracks[1];

/** An GPUExpEvaluator object to compute exponentials */
__constant__ GPUExpEvaluator exp_evaluator;


/**
 * @brief A struct used to check if a value on the GPU is equal to INF.
 * @details This is used as a predicate in Thrust routines.
 */
struct isinf_test {
  /**
   * @brief Checks if a double precision value is INF.
   * @param a the value to check
   * @return true if equal to INF, false otherwise
   */
  __host__ __device__ bool operator()(double a) {
    return isinf(a);
  }

  /**
   * @brief Checks if a single precision value is INF.
   * @param a the value to check
   * @return true if equal to INF, false otherwise
   */
  __host__ __device__ bool operator()(float a) {
    return isinf(a);
  }
};


/**
 * @brief A struct used to check if a value on the GPU is equal to NaN.
 * @details This is used as a predicate in Thrust routines.
 */
struct isnan_test {
  /**
   * @brief Checks if a double precision value is NaN.
   * @param a the value to check
   * @return true if equal to NaN, false otherwise
   */
  __host__ __device__ bool operator()(double a) {
    return isnan(a);
  }

  /**
   * @brief Checks if a single precision value is NaN.
   * @param a the value to check
   * @return true if equal to NaN, false otherwise
   */
  __host__ __device__ bool operator()(float a) {
    return isnan(a);
  }
};


/**
 * @brief A functor to multiply all elements in a Thrust vector by a constant.
 * @param constant the constant to multiply the vector
 */
template< typename T >
struct multiplyByConstant {

public:
  /* The constant to multiply by */
  const T constant;

  /**
   * @brief Constructor for the functor.
   * @param constant to multiply each element in a Thrust vector
   */
  multiplyByConstant(T constant) : constant(constant) {}

  /**
   * @brief Multiply an element in a Thrust vector.
   * @param VecElem the element to multiply
   */
  __host__ __device__ void operator()(T& VecElem) const {
    VecElem = VecElem * constant;
  }
};


/**
 * @class This provides a templated interface for a strided iterator over
 *        a Thrust device_vector on a GPU.
 * @details This code is taken from the Thrust examples site on 1/20/2015:
 *           https://github.com/thrust/thrust/blob/master/examples/strided_range.cu
 */
template <typename Iterator>
class strided_range {

public:

  typedef typename thrust::iterator_difference<Iterator>::type difference_type;

  struct stride_functor : public thrust::unary_function<difference_type,difference_type> {

    difference_type stride;

    stride_functor(difference_type stride) : stride(stride) { }

    __host__ __device__ difference_type operator()(const difference_type& i) const {
      return stride * i;
    }
  };

  typedef typename thrust::counting_iterator<difference_type> CountingIterator;
  typedef typename thrust::transform_iterator<stride_functor, CountingIterator>
    TransformIterator;
  typedef typename thrust::permutation_iterator<Iterator,TransformIterator>
    PermutationIterator;
  typedef PermutationIterator iterator;

  /**
   * @brief The strided iterator constructor.
   */
  strided_range(Iterator first, Iterator last, difference_type stride)
    : first(first), last(last), stride(stride) { }

  /**
   * @brief Get the first element in the iterator.
   * @return the first element in the iterator
   */
  iterator begin(void) const {
    return PermutationIterator(first,
      TransformIterator(CountingIterator(0), stride_functor(stride)));
  }

  /**
   * @brief Get the last element in the iterator.
   * @return the last element in the iterator
   */
  iterator end(void) const {
    return begin() + ((last - first) + (stride - 1)) / stride;
  }

protected:

  /** The first element in the underlying device_vector as set by the constructor */
  Iterator first;

  /** The last element in the underlying device_vector as set by the constructor */
  Iterator last;

  /** The stride to use when iterating over the underlying device_vector */
  difference_type stride;

};


/**
 * @brief Compute the total fission source from all FSRs.
 * @param FSR_volumes an array of FSR volumes
 * @param FSR_materials an array of FSR Material indices
 * @param materials an array of dev_materials on the device
 * @param scalar_flux the scalar flux in each FSR and energy group
 * @param fission_sources array of fission sources in each FSR and energy group
 */
__global__ void computeFissionSourcesOnDevice(FP_PRECISION* FSR_volumes,
                                              int* FSR_materials,
                                              dev_material* materials,
                                              FP_PRECISION* scalar_flux,
                                              FP_PRECISION* fission_sources) {

  /* Use a shared memory buffer for each thread's fission source */
  extern __shared__ FP_PRECISION shared_fission_source[];

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  dev_material* curr_material;
  FP_PRECISION* nu_sigma_f;
  FP_PRECISION volume, source;

  /* Initialize fission source to zero */
  shared_fission_source[threadIdx.x] = 0;

  /* Iterate over all FSRs */
  while (tid < *num_FSRs) {

    curr_material = &materials[FSR_materials[tid]];
    nu_sigma_f = curr_material->_nu_sigma_f;
    volume = FSR_volumes[tid];

    /* Iterate over energy groups and update fission source for
     * this thread block */
    for (int e=0; e < *num_groups; e++) {
      source = nu_sigma_f[e] * scalar_flux(tid,e) * volume;
      shared_fission_source[threadIdx.x] += source;
    }

    /* Increment thread id */
    tid += blockDim.x * gridDim.x;
  }

  /* Copy this thread's fission source to global memory */
  tid = threadIdx.x + blockIdx.x * blockDim.x;
  fission_sources[tid] = shared_fission_source[threadIdx.x];
}


/**
 * @brief Computes the total source (fission, scattering, fixed) in each FSR.
 * @details This method computes the total source in each region based on
 *          this iteration's current approximation to the scalar flux.
 * @param FSR_materials an array of FSR Material indices
 * @param materials an array of dev_material pointers
 * @param scalar_flux an array of FSR scalar fluxes
 * @param fixed_sources an array of fixed (user-defined) sources
 * @param reduced_sources an array of FSR sources / total xs
 * @param inverse_k_eff the inverse of keff
 */
__global__ void computeFSRSourcesOnDevice(int* FSR_materials,
                                          dev_material* materials,
                                          FP_PRECISION* scalar_flux,
                                          FP_PRECISION* fixed_sources,
                                          FP_PRECISION* reduced_sources,
                                          FP_PRECISION inverse_k_eff) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  FP_PRECISION fission_source;
  FP_PRECISION scatter_source;

  dev_material* curr_material;
  FP_PRECISION* sigma_t;
  FP_PRECISION* sigma_s;
  FP_PRECISION* fiss_mat;

  /* Iterate over all FSRs */
  while (tid < *num_FSRs) {

    curr_material = &materials[FSR_materials[tid]];

    sigma_t = curr_material->_sigma_t;
    sigma_s = curr_material->_sigma_s;
    fiss_mat = curr_material->_fiss_matrix;

    /* Compute scatter + fission source for group g */
    for (int g=0; g < *num_groups; g++) {
      scatter_source = 0;
      fission_source = 0;

      for (int g_prime=0; g_prime < *num_groups; g_prime++) {
        scatter_source += sigma_s[g*(*num_groups)+g_prime] * scalar_flux(tid,g_prime);
        fission_source += fiss_mat[g*(*num_groups)+g_prime] * scalar_flux(tid,g_prime);
      }

      fission_source *= inverse_k_eff;

      /* Compute total (scatter+fission+fixed) reduced source */
      reduced_sources(tid,g) = fixed_sources(tid,g);
      reduced_sources(tid,g) += scatter_source + fission_source;
      reduced_sources(tid,g) *= ONE_OVER_FOUR_PI;
      reduced_sources(tid,g) = __fdividef(reduced_sources(tid,g), sigma_t[g]);
    }

    /* Increment the thread id */
    tid += blockDim.x * gridDim.x;
  }
}


/**
 * @brief Computes the total fission source in each FSR in each energy group
 * @details This method is a helper routine for the openmoc.krylov submodule.
 *          This routine computes the total fission source in each FSR. If the
 *          divide_sigma_t parameter is true then the fission source will be
 *          divided by the total cross-section in each FSR.
 * @param FSR_materials an array of FSR Material indices
 * @param materials an array of dev_material pointers
 * @param divide_sigma_t a boolean indicating whether to divide by the total xs
 * @param scalar_flux an array of FSR scalar fluxes
 * @param reduced_sources an array of FSR fission sources
 */
__global__ void computeFSRFissionSourcesOnDevice(int* FSR_materials,
                                                 dev_material* materials,
						 bool divide_sigma_t,
                                                 FP_PRECISION* scalar_flux,
                                                 FP_PRECISION* reduced_sources) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  FP_PRECISION fission_source;

  dev_material* curr_material;
  FP_PRECISION* sigma_t;
  FP_PRECISION* fiss_mat;

  /* Iterate over all FSRs */
  while (tid < *num_FSRs) {

    curr_material = &materials[FSR_materials[tid]];

    sigma_t = curr_material->_sigma_t;
    fiss_mat = curr_material->_fiss_matrix;

    /* Compute fission source for group g */
    for (int g=0; g < *num_groups; g++) {
      fission_source = 0;

      for (int g_prime=0; g_prime < *num_groups; g_prime++)
        fission_source += fiss_mat[g*(*num_groups)+g_prime] * scalar_flux(tid,g_prime);

      /* Set the reduced fission source for FSR tid in group g */
      reduced_sources(tid,g) = fission_source;
      reduced_sources(tid,g) *= ONE_OVER_FOUR_PI;
      if (divide_sigma_t)
        reduced_sources(tid,g) = __fdividef(reduced_sources(tid,g), sigma_t[g]);
    }

    /* Increment the thread id */
    tid += blockDim.x * gridDim.x;
  }
}


/**
 * @brief Computes the total scattering source in each FSR and energy group.
 * @details This method is a helper routine for the openmoc.krylov submodule.
 *          This routine computes the total scatter source in each FSR. If the
 *          divide_sigma_t parameter is true then the scatter source will be
 *          divided by the total cross-section in each FSR.
 * @param FSR_materials an array of FSR Material indices
 * @param materials an array of dev_material pointers
 * @param divide_sigma_t a boolean indicating whether to divide by the total xs
 * @param scalar_flux an array of FSR scalar fluxes
 * @param reduced_sources an array of FSR scatter sources
 */
__global__ void computeFSRScatterSourcesOnDevice(int* FSR_materials,
                                                 dev_material* materials,
						 bool divide_sigma_t,
                                                 FP_PRECISION* scalar_flux,
                                                 FP_PRECISION* reduced_sources) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  FP_PRECISION scatter_source;

  dev_material* curr_material;
  FP_PRECISION* sigma_s;
  FP_PRECISION* sigma_t;

  /* Iterate over all FSRs */
  while (tid < *num_FSRs) {

    curr_material = &materials[FSR_materials[tid]];

    sigma_s = curr_material->_sigma_s;
    sigma_t = curr_material->_sigma_t;

    /* Compute total scattering source for this FSR in group g */
    for (int g=0; g < *num_groups; g++) {
      scatter_source = 0;

      for (int g_prime=0; g_prime < *num_groups; g_prime++)
        scatter_source += sigma_s[g*(*num_groups)+g_prime] * scalar_flux(tid,g_prime);

      /* Set the reduced scatter source for FSR tid in group g */
      reduced_sources(tid,g) = scatter_source;
      reduced_sources(tid,g) *= ONE_OVER_FOUR_PI;
      if (divide_sigma_t)
        reduced_sources(tid,g) = __fdividef(reduced_sources(tid,g), sigma_t[g]);
    }

    /* Increment the thread id */
    tid += blockDim.x * gridDim.x;
  }
}


/**
 * @brief Perform an atomic addition in double precision to an array address.
 * @details This method is straight out of CUDA C Developers Guide (cc 2013).
 * @param address the array memory address
 * @param val the value to add to the array
 * @return the atomically added array value and input value
 */
__device__ double atomicAdd(double* address, double val) {

  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +
                    __longlong_as_double(assumed)));
  } while (assumed != old);

  return __longlong_as_double(old);
}


/**
 * @brief Computes the contribution to the FSR scalar flux from a Track
 *        segment in a single energy group.
 * @details This method integrates the angular flux for a Track segment across
 *        energy groups and polar angles, and tallies it into the FSR scalar
 *        flux, and updates the Track's angular flux.
 * @param curr_segment a pointer to the Track segment of interest
 * @param azim_index a pointer to the azimuthal angle index for this segment
 * @param energy_group the energy group of interest
 * @param materials the array of dev_material pointers
 * @param track_flux a pointer to the Track's angular flux
 * @param reduced_sources the array of FSR sources / total xs
 * @param scalar_flux the array of FSR scalar fluxes
 */
__device__ void tallyScalarFlux(dev_segment* curr_segment,
                                int azim_index,
                                int energy_group,
                                dev_material* materials,
                                FP_PRECISION* track_flux,
                                FP_PRECISION* reduced_sources,
                                FP_PRECISION* scalar_flux) {

  int fsr_id = curr_segment->_region_uid;
  FP_PRECISION length = curr_segment->_length;
  dev_material* curr_material = &materials[curr_segment->_material_index];
  FP_PRECISION* sigma_t = curr_material->_sigma_t;

  /* The change in angular flux long this Track segment in this FSR */
  FP_PRECISION delta_psi;
  FP_PRECISION exponential;

  /* Zero the FSR scalar flux contribution from this segment and energy group */
  FP_PRECISION fsr_flux = 0.0;

  /* Loop over polar angles */
  for (int p=0; p < *num_polar_2; p++) {
    exponential =
      exp_evaluator.computeExponential(sigma_t[energy_group] * length, p);
    delta_psi = (track_flux[p] - reduced_sources(fsr_id,energy_group));
    delta_psi *= exponential;
    fsr_flux += delta_psi * weights(azim_index,p);
    track_flux[p] -= delta_psi;
  }

  /* Atomically increment the scalar flux for this FSR */
  atomicAdd(&scalar_flux(fsr_id,energy_group), fsr_flux);
}


/**
 * @brief Updates the boundary flux for a Track given boundary conditions.
 * @details For reflective and periodic boundary conditions, the outgoing
 *          boundary flux for the Track is given to the corresponding reflecting
 *          or periodic Track. For vacuum boundary conditions, the outgoing flux
 *          is tallied as leakage. Note: Only one energy group is transferred
 *          by this routine.
 * @param curr_track a pointer to the Track of interest
 * @param azim_index a pointer to the azimuthal angle index for this segment
 * @param track_flux an array of the outgoing Track flux
 * @param boundary_flux an array of all angular fluxes
 * @param weights an array of Quadrature weights
 * @param energy_angle_index the energy group index
 * @param direction the Track direction (forward - true, reverse - false)
 */
__device__ void transferBoundaryFlux(dev_track* curr_track,
                                     int azim_index,
                                     FP_PRECISION* track_flux,
                                     FP_PRECISION* boundary_flux,
                                     int energy_angle_index,
                                     bool direction) {

  int start = energy_angle_index;
  bool transfer_flux;
  int track_out_id;

  /* For the "forward" direction */
  if (direction) {
    transfer_flux = curr_track->_transfer_flux_out;
    track_out_id = curr_track->_track_out;
    start += curr_track->_next_out * (*polar_times_groups);
  }

  /* For the "reverse" direction */
  else {
    transfer_flux = curr_track->_transfer_flux_in;
    track_out_id = curr_track->_track_in;
    start += curr_track->_next_in * (*polar_times_groups);
  }

  FP_PRECISION* track_out_flux = &boundary_flux(track_out_id,start);

  /* Put Track's flux in the shared memory temporary flux array */
  for (int p=0; p < *num_polar_2; p++)
    track_out_flux[p] = track_flux[p] * transfer_flux;
}


/**
 * @brief This method performs one transport sweep of one halfspace of all
 *        azimuthal angles, tracks, segments, polar angles and energy groups.
 * @details The method integrates the flux along each track and updates the
 *          boundary fluxes for the corresponding output Track, while updating
 *          the scalar flux in each FSR.
 * @param scalar_flux an array of FSR scalar fluxes
 * @param boundary_flux an array of Track boundary fluxes
 * @param reduced_sources an array of FSR sources / total xs
 * @param materials an array of dev_material pointers
 * @param tracks an array of Tracks
 * @param tid_offset the Track offset for azimuthal angle halfspace
 * @param tid_max the upper bound on the Track IDs for this azimuthal
 *                angle halfspace
 */
__global__ void transportSweepOnDevice(FP_PRECISION* scalar_flux,
                                       FP_PRECISION* boundary_flux,
                                       FP_PRECISION* reduced_sources,
                                       dev_material* materials,
                                       dev_track* tracks,
                                       int tid_offset,
                                       int tid_max) {

  /* Shared memory buffer for each thread's angular flux */
  extern __shared__ FP_PRECISION temp_flux[];
  FP_PRECISION* track_flux;

  int tid = tid_offset + threadIdx.x + blockIdx.x * blockDim.x;
  int track_id = tid / *num_groups;
  int track_flux_index = threadIdx.x * (*num_polar);
  int energy_group = tid % (*num_groups);
  int energy_angle_index = energy_group * (*num_polar_2);

  dev_track* curr_track;
  int azim_index;
  int num_segments;
  dev_segment* curr_segment;

  /* Iterate over Track with azimuthal angles in (0, pi/2) */
  while (track_id < tid_max) {

    /* Initialize local registers with important data */
    curr_track = &tracks[track_id];
    azim_index = curr_track->_azim_angle_index;
    num_segments = curr_track->_num_segments;

    /* Retrieve pointer to thread's shared memory buffer for angular flux */
    track_flux = &temp_flux[track_flux_index];

    /* Put Track's flux in the shared memory temporary flux array */
    for (int p=0; p < *num_polar_2; p++) {

      /* Forward flux along this Track */
      track_flux[p] = boundary_flux(track_id,p+energy_angle_index);

      /* Reverse flux along this Track */
      track_flux[(*num_polar_2) + p] =
            boundary_flux(track_id,p+energy_angle_index+(*polar_times_groups));
    }

    /* Loop over each Track segment in forward direction */
    for (int i=0; i < num_segments; i++) {
      curr_segment = &curr_track->_segments[i];
      tallyScalarFlux(curr_segment, azim_index, energy_group, materials,
                      track_flux, reduced_sources, scalar_flux);
    }

    /* Transfer boundary angular flux to outgoing Track */
    transferBoundaryFlux(curr_track, azim_index, track_flux, boundary_flux,
                         energy_angle_index, true);

    /* Loop over each Track segment in reverse direction */
    track_flux = &temp_flux[track_flux_index + (*num_polar_2)];

    for (int i=num_segments-1; i > -1; i--) {
      curr_segment = &curr_track->_segments[i];
      tallyScalarFlux(curr_segment, azim_index, energy_group, materials,
                      track_flux, reduced_sources, scalar_flux);
    }

    /* Transfer boundary angular flux to outgoing Track */
    transferBoundaryFlux(curr_track, azim_index, track_flux, boundary_flux,
                         energy_angle_index, false);

    /* Update the indices for this thread to the next Track, energy group */
    tid += blockDim.x * gridDim.x;
    track_id = tid / *num_groups;
    energy_group = tid % (*num_groups);
    energy_angle_index = energy_group * (*num_polar_2);
  }
}


/**
 * @brief Add the source term contribution in the transport equation to
 *        the FSR scalar flux on the GPU.
 * @param scalar_flux an array of FSR scalar fluxes
 * @param reduced_sources an array of FSR sources / total xs
 * @param FSR_volumes an array of FSR volumes
 * @param FSR_materials an array of FSR material indices
 * @param materials an array of dev_material pointers
 */
__global__ void addSourceToScalarFluxOnDevice(FP_PRECISION* scalar_flux,
                                              FP_PRECISION* reduced_sources,
                                              FP_PRECISION* FSR_volumes,
                                              int* FSR_materials,
                                              dev_material* materials) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  FP_PRECISION volume;

  dev_material* curr_material;
  FP_PRECISION* sigma_t;

  /* Iterate over all FSRs */
  while (tid < *num_FSRs) {

    curr_material = &materials[FSR_materials[tid]];
    volume = FSR_volumes[tid];
    sigma_t = curr_material->_sigma_t;

    /* Iterate over all energy groups */
    for (int i=0; i < *num_groups; i++) {
      scalar_flux(tid,i) = __fdividef(scalar_flux(tid,i),
                                     (sigma_t[i] * volume));
      scalar_flux(tid,i) += FOUR_PI * reduced_sources(tid,i);
    }

    /* Increment thread id */
    tid += blockDim.x * gridDim.x;
  }
}


/**
 * @brief Compute the total volume-intergrated fission source from
 *        all FSRs and energy groups.
 * @param FSR_volumes an array of the FSR volumes
 * @param FSR_materials an array of the FSR Material indices
 * @param materials an array of the dev_material pointers
 * @param scalar_flux an array of FSR scalar fluxes
 * @param fission an array of FSR nu-fission rates
 */
__global__ void computeFSRFissionRatesOnDevice(FP_PRECISION* FSR_volumes,
                                               int* FSR_materials,
                                               dev_material* materials,
                                               FP_PRECISION* scalar_flux,
                                               FP_PRECISION* fission) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  dev_material* curr_material;
  FP_PRECISION* nu_sigma_f;
  FP_PRECISION volume;

  FP_PRECISION fiss = 0.;

  /* Iterate over all FSRs */
  while (tid < *num_FSRs) {

    curr_material = &materials[FSR_materials[tid]];
    nu_sigma_f = curr_material->_nu_sigma_f;
    volume = FSR_volumes[tid];

    FP_PRECISION curr_fiss = 0.;

    /* Compute fission rates rates for this thread block */
    for (int e=0; e < *num_groups; e++)
      curr_fiss += nu_sigma_f[e] * scalar_flux(tid,e);

    fiss += curr_fiss * volume;

    /* Increment thread id */
    tid += blockDim.x * gridDim.x;
  }

  /* Copy this thread's fission to global memory */
  tid = threadIdx.x + blockIdx.x * blockDim.x;
  fission[tid] = fiss;
}


/**
 * @brief Constructor initializes arrays for dev_tracks and dev_materials..
 * @details The constructor initalizes the number of CUDA threads and thread
 *          blocks each to a default of 64.
 * @param track_generator an optional pointer to the TrackjGenerator
 */
GPUSolver::GPUSolver(TrackGenerator* track_generator) :

  Solver(track_generator) {

  /* The default number of thread blocks and threads per thread block */
  _B = 64;
  _T = 64;

  _materials = NULL;
  _dev_tracks = NULL;
  _FSR_materials = NULL;

  if (track_generator != NULL)
    setTrackGenerator(track_generator);
}


/**
 * @brief Solver destructor frees all memory on the device, including arrays
 *        for the FSR scalar fluxes and sources and Track boundary fluxes.
 */
GPUSolver::~GPUSolver() {

  if (_FSR_volumes != NULL) {
    hipFree(_FSR_volumes);
    _FSR_volumes = NULL;
  }

  if (_FSR_materials != NULL) {
    hipFree(_FSR_materials);
    _FSR_materials = NULL;
  }

  if (_materials != NULL) {
    hipFree(_materials);
    _materials = NULL;
  }

  if (_dev_tracks != NULL) {
    hipFree(_dev_tracks);
    _dev_tracks = NULL;
  }

  /* Clear Thrust vectors's memory on the device */
  _boundary_flux.clear();
  _scalar_flux.clear();
  _old_scalar_flux.clear();
  _fixed_sources.clear();
  _reduced_sources.clear();
}


/**
 * @brief Returns the number of thread blocks to execute on the GPU.
 * @return the number of thread blocks
 */
int GPUSolver::getNumThreadBlocks() {
  return _B;
}


/**
 * @brief Returns the number of threads per block to execute on the GPU.
 * @return the number of threads per block
 */
int GPUSolver::getNumThreadsPerBlock() {
  return _T;
}


/**
 * @brief Returns the source for some energy group for a flat source region
 * @details This is a helper routine used by the openmoc.process module.
 * @param fsr_id the ID for the FSR of interest
 * @param group the energy group of interest
 * @return the flat source region source
 */
FP_PRECISION GPUSolver::getFSRSource(int fsr_id, int group) {

  if (fsr_id >= _num_FSRs)
    log_printf(ERROR, "Unable to return a source for FSR ID = %d "
               "since the max FSR ID = %d", fsr_id, _num_FSRs-1);

  else if (fsr_id < 0)
    log_printf(ERROR, "Unable to return a source for FSR ID = %d "
               "since FSRs do not have negative IDs", fsr_id);

  else if (group-1 >= _num_groups)
    log_printf(ERROR, "Unable to return a source in group %d "
               "since there are only %d groups", group, _num_groups);

  else if (group <= 0)
    log_printf(ERROR, "Unable to return a source in group %d "
               "since groups must be greater or equal to 1", group);

  else if (_scalar_flux.size() == 0)
    log_printf(ERROR, "Unable to return a source "
               "since it has not yet been computed");

  /* Get host material */
  Material* host_material = _geometry->findFSRMaterial(fsr_id);

  /* Get cross sections and scalar flux */
  FP_PRECISION* sigma_s = host_material->getSigmaS();
  FP_PRECISION* fiss_mat = host_material->getFissionMatrix();

  FP_PRECISION* fsr_scalar_fluxes = new FP_PRECISION[_num_groups];
  FP_PRECISION* scalar_flux =
       thrust::raw_pointer_cast(&_scalar_flux[0]);
  hipMemcpy((void*)fsr_scalar_fluxes, (void*)&scalar_flux[fsr_id*_num_groups],
             _num_groups * sizeof(FP_PRECISION),
             hipMemcpyDeviceToHost);

  FP_PRECISION fission_source = 0.0;
  FP_PRECISION scatter_source = 0.0;
  FP_PRECISION total_source;

  /* Compute total scattering and fission sources for this FSR */
  for (int g=0; g < _num_groups; g++) {
    scatter_source += sigma_s[(group-1)*(_num_groups)+g]
                      * fsr_scalar_fluxes[g];
    fission_source += fiss_mat[(group-1)*(_num_groups)+g]
                      * fsr_scalar_fluxes[g];
  }

  fission_source /= _k_eff;

  /* Compute the total source */
  total_source = fission_source + scatter_source;

  /* Add in fixed source (if specified by user) */
  total_source += _fixed_sources(fsr_id,group-1);

  /* Normalize to solid angle for isotropic approximation */
  total_source *= ONE_OVER_FOUR_PI;

  delete [] fsr_scalar_fluxes;

  return total_source;
}


/**
 * @brief Returns the scalar flux for some FSR and energy group.
 * @param fsr_id the ID for the FSR of interest
 * @param group the energy group of interest
 * @return the FSR scalar flux
 */
FP_PRECISION GPUSolver::getFlux(int fsr_id, int group) {

  if (fsr_id >= _num_FSRs)
    log_printf(ERROR, "Unable to return a scalar flux for FSR ID = %d "
               "since the max FSR ID = %d", fsr_id, _num_FSRs-1);

  else if (fsr_id < 0)
    log_printf(ERROR, "Unable to return a scalar flux for FSR ID = %d "
               "since FSRs do not have negative IDs", fsr_id);

  else if (group-1 >= _num_groups)
    log_printf(ERROR, "Unable to return a scalar flux in group %d "
               "since there are only %d groups", group, _num_groups);

  else if (group <= 0)
    log_printf(ERROR, "Unable to return a scalar flux in group %d "
               "since groups must be greater or equal to 1", group);

  if (_scalar_flux.size() == 0)
    log_printf(ERROR, "Unable to return a scalar flux "
               "since it has not yet been computed");

  return _scalar_flux(fsr_id,group-1);
}


/**
 * @brief Fills an array with the scalar fluxes on the GPU.
 * @details This class method is a helper routine called by the OpenMOC
 *          Python "openmoc.krylov" module for Krylov subspace methods.
 *          Although this method appears to require two arguments, in
 *          reality it only requires one due to SWIG and would be called
 *          from within Python as follows:
 *
 * @code
 *          num_fluxes = num_groups * num_FSRs
 *          fluxes = solver.getFluxes(num_fluxes)
 * @endcode
 *
 * @param fluxes an array of FSR scalar fluxes in each energy group
 * @param num_fluxes the total number of FSR flux values
 */
void GPUSolver::getFluxes(FP_PRECISION* out_fluxes, int num_fluxes) {

  if (num_fluxes != _num_groups * _num_FSRs)
    log_printf(ERROR, "Unable to get FSR scalar fluxes since there are "
               "%d groups and %d FSRs which does not match the requested "
               "%d flux values", _num_groups, _num_FSRs, num_fluxes);

  else if (_scalar_flux.size() == 0)
    log_printf(ERROR, "Unable to get FSR scalar fluxes since they "
               "have not yet been allocated on the device");

  FP_PRECISION* scalar_flux =
       thrust::raw_pointer_cast(&_scalar_flux[0]);

  /* Copy the fluxes from the GPU to the input array */
  hipMemcpy((void*)out_fluxes, (void*)scalar_flux,
            num_fluxes * sizeof(FP_PRECISION), hipMemcpyDeviceToHost);
}


/**
 * @brief Sets the number of thread blocks (>0) for CUDA kernels.
 * @param num_blocks the number of thread blocks
 */
void GPUSolver::setNumThreadBlocks(int num_blocks) {

  if (num_blocks < 0)
    log_printf(ERROR, "Unable to set the number of CUDA thread blocks "
               "to %d since it is a negative number", num_blocks);

  _B = num_blocks;
}


/**
 * @brief Sets the number of threads per block (>0) for CUDA kernels.
 * @param num_threads the number of threads per block
 */
void GPUSolver::setNumThreadsPerBlock(int num_threads) {

  if (num_threads < 0)
    log_printf(ERROR, "Unable to set the number of CUDA threads per block "
               "to %d since it is a negative number", num_threads);

  _T = num_threads;
}


/**
 * @brief Sets the Geometry for the Solver.
 * @details This is a private setter method for the Solver and is not
 *          intended to be called by the user.
 * @param geometry a pointer to a Geometry object
 */
void GPUSolver::setGeometry(Geometry* geometry) {

  Solver::setGeometry(geometry);

  std::map<int, Material*> host_materials=_geometry->getAllMaterials();
  std::map<int, Material*>::iterator iter;
  int material_index = 0;

  /* Iterate through all Materials and clone them as dev_material structs
   * on the device */
  for (iter=host_materials.begin(); iter != host_materials.end(); ++iter) {
    _material_IDs_to_indices[iter->second->getId()] = material_index;
    material_index++;
  }
}


/**
 * @brief Sets the Solver's TrackGenerator with characteristic Tracks.
 * @details The TrackGenerator must already have generated Tracks and have
 *          used ray tracing to segmentize them across the Geometry. This
 *          should be initated in Python prior to assigning the TrackGenerator
 *          to the Solver:
 *
 * @code
 *          track_generator.generateTracks()
 *          solver.setTrackGenerator(track_generator)
 * @endcode
 *
 * @param track_generator a pointer to a TrackGenerator object
 */
void GPUSolver::setTrackGenerator(TrackGenerator* track_generator) {
  Solver::setTrackGenerator(track_generator);
  initializeTracks();
  copyQuadrature();
}


/**
 * @brief Set the flux array for use in transport sweep source calculations.
 * @detail This is a helper method for the checkpoint restart capabilities,
 *         as well as the IRAMSolver in the openmoc.krylov submodule. This
 *         routine may be used as follows from within Python:
 *
 * @code
 *          num_FSRs = solver.getGeometry.getNumFSRs()
 *          num_groups = solver.getGeometry.getNumEnergyGroups()
 *          fluxes = numpy.random.rand(num_FSRs * num_groups, dtype=np.float)
 *          solver.setFluxes(fluxes)
 * @endcode
 *
 *          NOTE: This routine stores a pointer to the fluxes for the Solver
 *          to use during transport sweeps and other calculations. Hence, the
 *          flux array pointer is shared between NumPy and the Solver.
 *
 * @param in_fluxes an array with the fluxes to use
 * @param num_fluxes the number of flux values (# groups x # FSRs)
 */
void GPUSolver::setFluxes(FP_PRECISION* in_fluxes, int num_fluxes) {
  if (num_fluxes != _num_groups * _num_FSRs)
    log_printf(ERROR, "Unable to set an array with %d flux values for %d "
               " groups and %d FSRs", num_fluxes, _num_groups, _num_FSRs);

  /* Allocate array if flux arrays have not yet been initialized */
  if (_scalar_flux.size() == 0)
    initializeFluxArrays();

  FP_PRECISION* scalar_flux =
       thrust::raw_pointer_cast(&_scalar_flux[0]);

  /* Copy the input fluxes onto the GPU */
  hipMemcpy((void*)scalar_flux, (void*)in_fluxes,
             num_fluxes * sizeof(FP_PRECISION), hipMemcpyHostToDevice);
  _user_fluxes = true;
}


/**
 * @brief Creates a polar quadrature object for the GPUSolver on the GPU.
 */
void GPUSolver::copyQuadrature() {

  log_printf(INFO, "Copying quadrature on the GPU...");

  if (_num_polar_2 > MAX_POLAR_ANGLES_GPU)
    log_printf(ERROR, "Unable to initialize a polar quadrature with %d "
               "angles for the GPUSolver which is limited to %d polar "
               "angles. Update the MAX_POLAR_ANGLES_GPU macro in constants.h "
               "and recompile.", _num_polar_2, MAX_POLAR_ANGLES_GPU);

  /* Copy half the number of polar angles to constant memory on the GPU */
  hipMemcpyToSymbol(HIP_SYMBOL(num_polar_2), (void*)&_num_polar_2, sizeof(int), 0,
                     hipMemcpyHostToDevice);

  /* Copy the number of polar angles to constant memory on the GPU */
  int n_polar = 2 * _num_polar_2;
  hipMemcpyToSymbol(HIP_SYMBOL(num_polar), (void*)&n_polar,
                     sizeof(int), 0, hipMemcpyHostToDevice);

  /* Copy the weights to constant memory on the GPU */
  int num_azim_2 = _quadrature->getNumAzimAngles() / 2;
  FP_PRECISION total_weights[num_azim_2 * _num_polar_2];
  for (int a=0; a < num_azim_2; a++)
    for (int p=0; p < _num_polar_2; p++)
      total_weights[a*_num_polar_2 + p] = _quadrature->getWeight(a, p);
  hipMemcpyToSymbol(HIP_SYMBOL(weights), (void*)total_weights,
      _num_polar_2 * num_azim_2 * sizeof(FP_PRECISION), 0, hipMemcpyHostToDevice);

  /* Copy the sines of the polar angles which is needed if the user
   * requested the use of the exp intrinsic to evaluate exponentials */
  hipMemcpyToSymbol(HIP_SYMBOL(sin_thetas), (void*)_quadrature->getSinThetas(),
                     _num_polar_2 * sizeof(FP_PRECISION), 0,
                     hipMemcpyHostToDevice);
}


/**
 * @brief Initializes new GPUExpEvaluator object to compute exponentials.
 */
void GPUSolver::initializeExpEvaluator() {

  Solver::initializeExpEvaluator();

  log_printf(INFO, "Initializing the exponential evaluator on the GPU...");

  /* Allocate memory for a GPUExpEvaluator on the device */
  GPUExpEvaluator* dev_exp_evaluator;
  hipMalloc((void**)&dev_exp_evaluator, sizeof(GPUExpEvaluator));

  /* Clone ExpEvaluator from the host into GPUExpEvaluator on the device */
  clone_exp_evaluator(_exp_evaluator, dev_exp_evaluator);

  /* Copy the GPUExpEvaluator into constant memory on the GPU */
  hipMemcpyToSymbol(HIP_SYMBOL(exp_evaluator), (void*)dev_exp_evaluator,
                     sizeof(GPUExpEvaluator), 0, hipMemcpyDeviceToDevice);
}


/**
 * @brief Initializes the FSR volumes and dev_materials array on the GPU.
 * @details This method assigns each FSR a unique, monotonically increasing
 *          ID, sets the Material for each FSR, and assigns a volume based on
 *          the cumulative length of all of the segments inside the FSR.
 */
void GPUSolver::initializeFSRs() {

  log_printf(NORMAL, "Initializing FSRs on the GPU...");

  /* Delete old FSRs array if it exists */
  if (_FSR_volumes != NULL) {
    hipFree(_FSR_volumes);
    _FSR_volumes = NULL;
  }

  if (_FSR_materials != NULL) {
    hipFree(_FSR_materials);
    _FSR_materials = NULL;
  }

  Solver::initializeFSRs();

  /* Allocate memory for all FSR volumes and dev_materials on the device */
  try{

    /* Store pointers to arrays of FSR data created on the host by the
     * the parent class Solver::initializeFSRs() routine */
    FP_PRECISION* host_FSR_volumes = _FSR_volumes;
    int* host_FSR_materials = _FSR_materials;

    /* Allocate memory on device for FSR volumes and Material indices */
    hipMalloc((void**)&_FSR_volumes, _num_FSRs * sizeof(FP_PRECISION));
    hipMalloc((void**)&_FSR_materials, _num_FSRs * sizeof(int));

    /* Create a temporary FSR to material indices array */
    int* FSRs_to_material_indices = new int[_num_FSRs];

    /* Populate FSR Material indices array */
    for (int i = 0; i < _num_FSRs; i++)
      FSRs_to_material_indices[i] = _material_IDs_to_indices[_geometry->
        findFSRMaterial(i)->getId()];

    /* Copy the arrays of FSR data to the device */
    hipMemcpy((void*)_FSR_volumes, (void*)host_FSR_volumes,
      _num_FSRs * sizeof(FP_PRECISION), hipMemcpyHostToDevice);
    hipMemcpy((void*)_FSR_materials, (void*)FSRs_to_material_indices,
      _num_FSRs * sizeof(int), hipMemcpyHostToDevice);

    /* Copy the number of FSRs into constant memory on the GPU */
    hipMemcpyToSymbol(HIP_SYMBOL(num_FSRs), (void*)&_num_FSRs, sizeof(int), 0,
      hipMemcpyHostToDevice);

    /* Free the array of FSRs data allocated by the Solver parent class */
    free(host_FSR_materials);

    /* Free the temporary array of FSRs to material indices on the host */
    free(FSRs_to_material_indices);
  }
  catch(std::exception &e) {
    log_printf(ERROR, "Could not allocate memory for FSRs on GPU");
  }
}


/**
 * @brief Allocates all Materials data on the GPU.
 * @details This method loops over the materials in the host_materials map.
 *          Since CUDA does not support std::map data types on the device,
 *          the materials map must be converted to an array and a map created
 *          that maps a material ID to an indice in the new materials array. In
 *          initializeTracks, this map is used to convert the Material ID
 *          associated with every segment to an index in the materials array.
 * @param mode the solution type (FORWARD or ADJOINT)
 */
void GPUSolver::initializeMaterials(solverMode mode) {

  Solver::initializeMaterials(mode);

  log_printf(INFO, "Initializing materials on the GPU...");

  /* Copy the number of energy groups to constant memory on the GPU */
  hipMemcpyToSymbol(HIP_SYMBOL(num_groups), (void*)&_num_groups, sizeof(int), 0,
                     hipMemcpyHostToDevice);

  /* Copy the number of polar angles times energy groups to constant memory
   * on the GPU */
  hipMemcpyToSymbol(HIP_SYMBOL(polar_times_groups), (void*)&_polar_times_groups,
                     sizeof(int), 0, hipMemcpyHostToDevice);

  /* Delete old materials array if it exists */
  if (_materials != NULL)
    hipFree(_materials);

  /* Allocate memory for all dev_materials on the device */
  try{

    std::map<int, Material*> host_materials=_geometry->getAllMaterials();
    std::map<int, Material*>::iterator iter;
    int material_index = 0;

    /* Iterate through all Materials and clone them as dev_material structs
     * on the device */
    hipMalloc((void**)&_materials, _num_materials * sizeof(dev_material));
    for (iter=host_materials.begin(); iter != host_materials.end(); ++iter) {
      clone_material(iter->second, &_materials[material_index]);
      material_index++;
    }
  }
  catch(std::exception &e) {
    log_printf(ERROR, "Could not allocate memory for Materials on GPU");
  }
}


/**
 * @brief Allocates memory for all Tracks on the GPU
 */
void GPUSolver::initializeTracks() {

  log_printf(INFO, "Initializing tracks on the GPU...");

  /* Delete old Tracks array if it exists */
  if (_dev_tracks != NULL)
    hipFree(_dev_tracks);

  /* Allocate memory for all Tracks and Track offset indices on the device */
  try{

    /* Allocate array of dev_tracks */
    hipMalloc((void**)&_dev_tracks, _tot_num_tracks * sizeof(dev_track));

    /* Iterate through all Tracks and clone them as dev_tracks on the device */
    int index;

    for (int i=0; i < _tot_num_tracks; i++) {

      clone_track(_tracks[i], &_dev_tracks[i], _material_IDs_to_indices);

      /* Get indices to next tracks along "forward" and "reverse" directions */
      index = _tracks[i]->getTrackIn()->getUid();
      hipMemcpy((void*)&_dev_tracks[i]._track_in,
                 (void*)&index, sizeof(int), hipMemcpyHostToDevice);

      index = _tracks[i]->getTrackOut()->getUid();
      hipMemcpy((void*)&_dev_tracks[i]._track_out,
                 (void*)&index, sizeof(int), hipMemcpyHostToDevice);
    }

    /* Copy the total number of Tracks into constant memory on GPU */
    hipMemcpyToSymbol(HIP_SYMBOL(tot_num_tracks), (void*)&_tot_num_tracks,
                       sizeof(int), 0, hipMemcpyHostToDevice);
  }

  catch(std::exception &e) {
    log_printf(ERROR, "Could not allocate memory for Tracks on GPU");
  }
}


/**
 * @brief Allocates memory for Track boundary angular and FSR scalar fluxes.
 * @details Deletes memory for old flux vectors if they were allocated for a
 *          previous simulation.
 */
void GPUSolver::initializeFluxArrays() {

  log_printf(INFO, "Initializing flux vectors on the GPU...");

  /* Clear Thrust vectors' memory if previously allocated */
  _boundary_flux.clear();
  _scalar_flux.clear();
  _old_scalar_flux.clear();

  /* Allocate memory for all flux arrays on the device */
  try {
    int size = 2 * _tot_num_tracks * _polar_times_groups;
    _boundary_flux.resize(size);

    size = _num_FSRs * _num_groups;
    _scalar_flux.resize(size);
    _old_scalar_flux.resize(size);
  }
  catch(std::exception &e) {
    log_printf(ERROR, "Could not allocate memory for fluxes on GPU");
  }
}


/**
 * @brief Allocates memory for FSR source vectors on the GPU.
 * @details Deletes memory for old source vectors if they were allocated
 *          for a previous simulation.
 */
void GPUSolver::initializeSourceArrays() {

  log_printf(INFO, "Initializing source vectors on the GPU...");

  /* Clear Thrust vectors' memory if previously allocated */
  _reduced_sources.clear();
  _fixed_sources.clear();

  int size = _num_FSRs * _num_groups;

  /* Allocate memory for all source arrays on the device */
  try{
    _reduced_sources.resize(size);
    _fixed_sources.resize(size);
  }
  catch(std::exception &e) {
    log_printf(ERROR, "Could not allocate memory for sources on GPU");
  }

  /* Initialize fixed sources to zero */
  thrust::fill(_fixed_sources.begin(), _fixed_sources.end(), 0.0);

  /* Fill fixed sources with those assigned by Cell, Material or FSR */
  initializeFixedSources();
}


/**
 * @brief Populates array of fixed sources assigned by FSR.
 */
void GPUSolver::initializeFixedSources() {

  Solver::initializeFixedSources();

  int fsr_id, group;
  std::pair<int, int> fsr_group_key;
  std::map< std::pair<int, int>, FP_PRECISION >::iterator fsr_iter;

  /* Populate fixed source array with any user-defined sources */
  for (fsr_iter = _fix_src_FSR_map.begin();
       fsr_iter != _fix_src_FSR_map.end(); ++fsr_iter) {

    /* Get the FSR with an assigned fixed source */
    fsr_group_key = fsr_iter->first;
    fsr_id = fsr_group_key.first;
    group = fsr_group_key.second;

    if (group <= 0 || group > _num_groups)
      log_printf(ERROR,"Unable to use fixed source for group %d in "
                 "a %d energy group problem", group, _num_groups);

    if (fsr_id < 0 || fsr_id >= _num_FSRs)
      log_printf(ERROR,"Unable to use fixed source for FSR %d with only "
                 "%d FSRs in the geometry", fsr_id, _num_FSRs);

    _fixed_sources(fsr_id, group-1) = _fix_src_FSR_map[fsr_group_key];
  }
}


/**
 * @brief Zero each Track's boundary fluxes for each energy group and polar
 *        angle in the "forward" and "reverse" directions.
 */
void GPUSolver::zeroTrackFluxes() {
  thrust::fill(_boundary_flux.begin(), _boundary_flux.end(), 0.0);
}


/**
 * @brief Set the scalar flux for each FSR and energy group to some value.
 * @param value the value to assign to each FSR scalar flux
 */
void GPUSolver::flattenFSRFluxes(FP_PRECISION value) {
  thrust::fill(_scalar_flux.begin(), _scalar_flux.end(), value);
}


/**
 * @brief Stores the FSR scalar fluxes in the old scalar flux array.
 */
void GPUSolver::storeFSRFluxes() {
  thrust::copy(_scalar_flux.begin(), _scalar_flux.end(),
               _old_scalar_flux.begin());
}


/**
 * @brief Normalizes all FSR scalar fluxes and Track boundary angular
 *        fluxes to the total fission source (times \f$ \nu \f$).
 */
void GPUSolver::normalizeFluxes() {

  /** Create Thrust vector of fission sources in each FSR */
  thrust::device_vector<FP_PRECISION> fission_sources_vec;
  fission_sources_vec.resize(_B * _T);
  FP_PRECISION* fission_sources =
       thrust::raw_pointer_cast(&fission_sources_vec[0]);

  FP_PRECISION* scalar_flux =
       thrust::raw_pointer_cast(&_scalar_flux[0]);

  int shared_mem = sizeof(FP_PRECISION) * _T;

  computeFissionSourcesOnDevice<<<_B, _T, shared_mem>>>(_FSR_volumes,
                                                        _FSR_materials,
                                                        _materials,
                                                        scalar_flux,
                                                        fission_sources);

  FP_PRECISION norm_factor = 1.0 / thrust::reduce(fission_sources_vec.begin(),
                                                  fission_sources_vec.end());

  /* Multiply all scalar and angular fluxes by the normalization constant */
  thrust::transform(_scalar_flux.begin(), _scalar_flux.end(),
                    thrust::constant_iterator<FP_PRECISION>(norm_factor),
                    _scalar_flux.begin(), thrust::multiplies<FP_PRECISION>());
  thrust::transform(_old_scalar_flux.begin(), _old_scalar_flux.end(),
                    thrust::constant_iterator<FP_PRECISION>(norm_factor),
                    _old_scalar_flux.begin(),
                    thrust::multiplies<FP_PRECISION>());
  thrust::transform(_boundary_flux.begin(), _boundary_flux.end(),
                    thrust::constant_iterator<FP_PRECISION>(norm_factor),
                    _boundary_flux.begin(), thrust::multiplies<FP_PRECISION>());

  /* Clear Thrust vector of FSR fission sources */
  fission_sources_vec.clear();
}


/**
 * @brief Computes the total source (fission, scattering, fixed) in each FSR.
 * @details This method computes the total source in each FSR based on
 *          this iteration's current approximation to the scalar flux.
 */
void GPUSolver::computeFSRSources() {

  FP_PRECISION* scalar_flux =
       thrust::raw_pointer_cast(&_scalar_flux[0]);
  FP_PRECISION* fixed_sources =
       thrust::raw_pointer_cast(&_fixed_sources[0]);
  FP_PRECISION* reduced_sources =
       thrust::raw_pointer_cast(&_reduced_sources[0]);

  computeFSRSourcesOnDevice<<<_B, _T>>>(_FSR_materials, _materials,
                                        scalar_flux, fixed_sources,
                                        reduced_sources, 1.0 / _k_eff);
}


/**
 * @brief Computes the fission source in each FSR.
 * @details This method computes the fission source in each FSR based on
 *          this iteration's current approximation to the scalar flux.
 */
void GPUSolver::computeFSRFissionSources() {

  FP_PRECISION* scalar_flux =
       thrust::raw_pointer_cast(&_scalar_flux[0]);
  FP_PRECISION* reduced_sources =
       thrust::raw_pointer_cast(&_reduced_sources[0]);

  computeFSRFissionSourcesOnDevice<<<_B, _T>>>(_FSR_materials, _materials, true,
                                               scalar_flux, reduced_sources);
}


/**
 * @brief Computes the scatter source in each FSR.
 * @details This method computes the scatter source in each FSR based on
 *          this iteration's current approximation to the scalar flux.
 */
void GPUSolver::computeFSRScatterSources() {

  FP_PRECISION* scalar_flux =
       thrust::raw_pointer_cast(&_scalar_flux[0]);
  FP_PRECISION* reduced_sources =
       thrust::raw_pointer_cast(&_reduced_sources[0]);

  computeFSRScatterSourcesOnDevice<<<_B, _T>>>(_FSR_materials, _materials, true,
                                               scalar_flux, reduced_sources);
}


/**
 * @brief This method performs one transport sweep of all azimuthal angles,
 *        Tracks, Track segments, polar angles and energy groups.
 * @details The method integrates the flux along each Track and updates the
 *          boundary fluxes for the corresponding output Track, while updating
 *          the scalar flux in each flat source region.
 */
void GPUSolver::transportSweep() {

  int shared_mem = _T * _num_polar_2 * 2 * sizeof(FP_PRECISION);
  int tid_offset = 0;
  int tid_max = 0;

  log_printf(DEBUG, "Transport sweep on device with %d blocks and %d threads",
             _B, _T);

  /* Get device pointer to the Thrust vectors */
  FP_PRECISION* scalar_flux =
       thrust::raw_pointer_cast(&_scalar_flux[0]);
  FP_PRECISION* boundary_flux =
       thrust::raw_pointer_cast(&_boundary_flux[0]);
  FP_PRECISION* reduced_sources =
       thrust::raw_pointer_cast(&_reduced_sources[0]);

  /* Initialize flux in each FSR to zero */
  flattenFSRFluxes(0.0);

  /* Loop over the parallel track groups and perform transport sweep on tracks
   * in that group */
  for (int g=0; g < _num_parallel_track_groups; g++) {
    tid_offset = tid_max * _num_groups;
    tid_max += _track_generator->getNumTracksByParallelGroup(g);

    transportSweepOnDevice<<<_B, _T, shared_mem>>>(scalar_flux, boundary_flux,
                                                   reduced_sources,
                                                   _materials, _dev_tracks,
                                                   tid_offset, tid_max);

    hipDeviceSynchronize();
  }
}


/**
 * @brief Add the source term contribution in the transport equation to
 *        the FSR scalar flux.
 */
void GPUSolver::addSourceToScalarFlux() {
  FP_PRECISION* scalar_flux =
       thrust::raw_pointer_cast(&_scalar_flux[0]);
  FP_PRECISION* reduced_sources =
       thrust::raw_pointer_cast(&_reduced_sources[0]);

  addSourceToScalarFluxOnDevice<<<_B,_T>>>(scalar_flux, reduced_sources,
                                           _FSR_volumes, _FSR_materials,
                                           _materials);
}


/**
 * @brief Compute \f$ k_{eff} \f$ from successive fission sources.
 * @details This method computes the current approximation to the
 *          multiplication factor on this iteration as follows:
 *          \f$ k_{eff} = \frac{\displaystyle\sum_{i \in I}
 *                        \displaystyle\sum_{g \in G} \nu \Sigma^F_g \Phi V_{i}}
 *                        {\displaystyle\sum_{i \in I}
 *                        \displaystyle\sum_{g \in G} (\Sigma^T_g \Phi V_{i} -
 *                        \Sigma^S_g \Phi V_{i} - L_{i,g})} \f$
 */
void GPUSolver::computeKeff() {

  FP_PRECISION fission;

  thrust::device_vector<FP_PRECISION> fission_vec;
  fission_vec.resize(_B * _T);

  FP_PRECISION* fiss_ptr = thrust::raw_pointer_cast(&fission_vec[0]);
  FP_PRECISION* flux = thrust::raw_pointer_cast(&_scalar_flux[0]);

  /* Compute the total, fission and scattering reaction rates on device.
   * This kernel stores partial rates in a Thrust vector with as many
   * entries as CUDAthreads executed by the kernel */
  computeFSRFissionRatesOnDevice<<<_B, _T>>>(_FSR_volumes, _FSR_materials,
                                             _materials, flux, fiss_ptr);

  /* Compute the total fission source */
  fission = thrust::reduce(fission_vec.begin(), fission_vec.end());

  _k_eff *= fission;

  fission_vec.clear();
}


/**
 * @brief Computes the residual between source/flux iterations.
 * @param res_type the type of residuals to compute
 *        (SCALAR_FLUX, FISSION_SOURCE, TOTAL_SOURCE)
 * @return the average residual in each flat source region
 */
double GPUSolver::computeResidual(residualType res_type) {

  int norm;
  double residual;
  isinf_test inf_test;
  isnan_test nan_test;

  /* Allocate Thrust vector for residuals in each FSR */
  thrust::device_vector<double> residuals(_num_FSRs);

  if (res_type == SCALAR_FLUX) {

    norm = _num_FSRs;

    /* Allocate Thrust vector for residuals */
    thrust::device_vector<FP_PRECISION> fp_residuals(_num_FSRs * _num_groups);
    thrust::device_vector<FP_PRECISION> FSR_fp_residuals(_num_FSRs);

    /* Compute the relative flux change in each FSR and group */
    thrust::transform(_scalar_flux.begin(), _scalar_flux.end(),
                      _old_scalar_flux.begin(), fp_residuals.begin(),
                      thrust::minus<FP_PRECISION>());
    thrust::transform(fp_residuals.begin(), fp_residuals.end(),
                      _old_scalar_flux.begin(), fp_residuals.begin(),
                      thrust::divides<FP_PRECISION>());

    /* Replace INF and NaN values (from divide by zero) with 0. */
    thrust::replace_if(fp_residuals.begin(), fp_residuals.end(), inf_test, 0);
    thrust::replace_if(fp_residuals.begin(), fp_residuals.end(), nan_test, 0);

    /* Square the residuals */
    thrust::transform(fp_residuals.begin(), fp_residuals.end(),
                      fp_residuals.begin(), fp_residuals.begin(),
                      thrust::multiplies<FP_PRECISION>());

    typedef thrust::device_vector<FP_PRECISION>::iterator Iterator;

    /* Reduce flux residuals across energy groups within each FSR */
    for (int e=0; e < _num_groups; e++) {
      strided_range<Iterator> strider(fp_residuals.begin() + e,
                                      fp_residuals.end(), _num_groups);
      thrust::transform(FSR_fp_residuals.begin(), FSR_fp_residuals.end(),
                        strider.begin(), FSR_fp_residuals.begin(),
                        thrust::plus<FP_PRECISION>());
    }

    /* Copy the FP_PRECISION residual to the double precision residual */
    thrust::copy(FSR_fp_residuals.begin(),
                 FSR_fp_residuals.end(), residuals.begin());

    /* Sum up the residuals */
    residual = thrust::reduce(residuals.begin(), residuals.end());

    /* Deallocate memory for Thrust vectors */
    fp_residuals.clear();
    FSR_fp_residuals.clear();
    residuals.clear();

    /* Normalize the residual */
    residual = sqrt(residual / norm);

    return residual;
  }

  else if (res_type == FISSION_SOURCE) {

    if (_num_fissionable_FSRs == 0)
      log_printf(ERROR, "The Solver is unable to compute a "
                 "FISSION_SOURCE residual without fissionable FSRs");

    norm = _num_fissionable_FSRs;

    /* Allocate Thrust vectors for fission sources in each FSR, group */
    thrust::device_vector<FP_PRECISION> new_fission_sources_vec(_num_FSRs * _num_groups);
    thrust::device_vector<FP_PRECISION> old_fission_sources_vec(_num_FSRs * _num_groups);

    /* Allocate Thrust vectors for energy-integrated fission sources in each FSR */
    thrust::device_vector<FP_PRECISION> FSR_old_fiss_src(_num_FSRs);
    thrust::device_vector<FP_PRECISION> FSR_new_fiss_src(_num_FSRs);

    /* Cast Thrust vectors as array pointers */
    FP_PRECISION* old_fission_sources =
         thrust::raw_pointer_cast(&old_fission_sources_vec[0]);
    FP_PRECISION* new_fission_sources =
         thrust::raw_pointer_cast(&new_fission_sources_vec[0]);
    FP_PRECISION* scalar_flux =
         thrust::raw_pointer_cast(&_scalar_flux[0]);
    FP_PRECISION* old_scalar_flux =
         thrust::raw_pointer_cast(&_old_scalar_flux[0]);

    /* Compute the old and new nu-fission sources in each FSR, group */
    computeFSRFissionSourcesOnDevice<<<_B, _T>>>(_FSR_materials, _materials, false,
                                                 old_scalar_flux, old_fission_sources);
    computeFSRFissionSourcesOnDevice<<<_B, _T>>>(_FSR_materials, _materials, false,
                                                 scalar_flux, new_fission_sources);

    typedef thrust::device_vector<FP_PRECISION>::iterator Iterator;

    /* Reduce nu-fission sources across energy groups within each FSR */
    for (int e=0; e < _num_groups; e++) {
      strided_range<Iterator> old_strider(old_fission_sources_vec.begin() + e,
                                          old_fission_sources_vec.end(), _num_groups);
      strided_range<Iterator> new_strider(new_fission_sources_vec.begin() + e,
                                          new_fission_sources_vec.end(), _num_groups);
      thrust::transform(FSR_old_fiss_src.begin(), FSR_old_fiss_src.end(),
                        old_strider.begin(), FSR_old_fiss_src.begin(),
                        thrust::plus<FP_PRECISION>());
      thrust::transform(FSR_new_fiss_src.begin(), FSR_new_fiss_src.end(),
                        new_strider.begin(), FSR_new_fiss_src.begin(),
                        thrust::plus<FP_PRECISION>());
    }

    /* Compute the relative nu-fission source change in each FSR */
    thrust::transform(FSR_new_fiss_src.begin(), FSR_new_fiss_src.end(),
                      FSR_old_fiss_src.begin(), residuals.begin(),
                      thrust::minus<FP_PRECISION>());
    thrust::transform(residuals.begin(), residuals.end(),
                      FSR_old_fiss_src.begin(), residuals.begin(),
                      thrust::divides<FP_PRECISION>());

    /* Deallocate memory for Thrust vectors */
    old_fission_sources_vec.clear();
    new_fission_sources_vec.clear();
    FSR_old_fiss_src.clear();
    FSR_new_fiss_src.clear();
  }

  else if (res_type == TOTAL_SOURCE) {

    norm = _num_FSRs;

    /* Allocate Thrust vectors for fission/scatter sources in each FSR, group */
    thrust::device_vector<FP_PRECISION> new_sources_vec(_num_FSRs * _num_groups);
    thrust::device_vector<FP_PRECISION> old_sources_vec(_num_FSRs * _num_groups);
    thrust::fill(new_sources_vec.begin(), new_sources_vec.end(), 0.0);
    thrust::fill(old_sources_vec.begin(), old_sources_vec.end(), 0.0);

    /* Allocate Thrust vectors for energy-integrated fission/scatter sources in each FSR */
    thrust::device_vector<FP_PRECISION> FSR_old_src(_num_FSRs);
    thrust::device_vector<FP_PRECISION> FSR_new_src(_num_FSRs);
    thrust::fill(FSR_old_src.begin(), FSR_old_src.end(), 0.);
    thrust::fill(FSR_new_src.begin(), FSR_new_src.end(), 0.);

    /* Cast Thrust vectors as array pointers */
    FP_PRECISION* old_sources =
         thrust::raw_pointer_cast(&old_sources_vec[0]);
    FP_PRECISION* new_sources =
         thrust::raw_pointer_cast(&new_sources_vec[0]);
    FP_PRECISION* scalar_flux =
         thrust::raw_pointer_cast(&_scalar_flux[0]);
    FP_PRECISION* old_scalar_flux =
         thrust::raw_pointer_cast(&_old_scalar_flux[0]);

    /* Compute nu-fission source */

    /* Compute the old and new nu-fission sources in each FSR, group */
    computeFSRFissionSourcesOnDevice<<<_B, _T>>>(_FSR_materials, _materials, false,
                                                 old_scalar_flux, old_sources);
    computeFSRFissionSourcesOnDevice<<<_B, _T>>>(_FSR_materials, _materials, false,
                                                 scalar_flux, new_sources);

    typedef thrust::device_vector<FP_PRECISION>::iterator Iterator;

    /* Reduce nu-fission sources across energy groups within each FSR */
    for (int e=0; e < _num_groups; e++) {
      strided_range<Iterator> old_strider(old_sources_vec.begin() + e,
                                          old_sources_vec.end(), _num_groups);
      strided_range<Iterator> new_strider(new_sources_vec.begin() + e,
                                          new_sources_vec.end(), _num_groups);
      thrust::transform(FSR_old_src.begin(), FSR_old_src.end(),
                        old_strider.begin(), FSR_old_src.begin(),
                        thrust::plus<FP_PRECISION>());
      thrust::transform(FSR_new_src.begin(), FSR_new_src.end(),
                        new_strider.begin(), FSR_new_src.begin(),
                        thrust::plus<FP_PRECISION>());
    }

    /* Multiply fission sources by inverse keff */
    thrust::for_each(FSR_new_src.begin(), FSR_new_src.end(),
                     multiplyByConstant<FP_PRECISION>(1. / _k_eff));
    thrust::for_each(FSR_old_src.begin(), FSR_old_src.end(),
                     multiplyByConstant<FP_PRECISION>(1. / _k_eff));

    /* Compute scatter source */

    /* Reset sources Thrust vectors to zero */
    thrust::fill(new_sources_vec.begin(), new_sources_vec.end(), 0.0);
    thrust::fill(old_sources_vec.begin(), old_sources_vec.end(), 0.0);

    /* Compute the old and new scattering sources in each FSR, group */
    computeFSRScatterSourcesOnDevice<<<_B, _T>>>(_FSR_materials, _materials, false,
                                                 old_scalar_flux, old_sources);
    computeFSRScatterSourcesOnDevice<<<_B, _T>>>(_FSR_materials, _materials, false,
                                                 scalar_flux, new_sources);

    /* Reduce scatter sources across energy groups within each FSR */
    for (int e=0; e < _num_groups; e++) {
      strided_range<Iterator> old_strider(old_sources_vec.begin() + e,
                                          old_sources_vec.end(), _num_groups);
      strided_range<Iterator> new_strider(new_sources_vec.begin() + e,
                                          new_sources_vec.end(), _num_groups);
      thrust::transform(FSR_old_src.begin(), FSR_old_src.end(),
                        old_strider.begin(), FSR_old_src.begin(),
                        thrust::plus<FP_PRECISION>());
      thrust::transform(FSR_new_src.begin(), FSR_new_src.end(),
                        new_strider.begin(), FSR_new_src.begin(),
                        thrust::plus<FP_PRECISION>());
    }

    /* Compute the relative total source change in each FSR */
    thrust::transform(FSR_new_src.begin(), FSR_new_src.end(),
                      FSR_old_src.begin(), residuals.begin(),
                      thrust::minus<FP_PRECISION>());
    thrust::transform(residuals.begin(), residuals.end(),
                      FSR_old_src.begin(), residuals.begin(),
                      thrust::divides<FP_PRECISION>());

    /* Deallocate memory for Thrust vectors */
    old_sources_vec.clear();
    new_sources_vec.clear();
    FSR_old_src.clear();
    FSR_new_src.clear();
  }

  /* Replace INF and NaN values (from divide by zero) with 0. */
  thrust::replace_if(residuals.begin(), residuals.end(), inf_test, 0);
  thrust::replace_if(residuals.begin(), residuals.end(), nan_test, 0);

  /* Square the residuals */
  thrust::transform(residuals.begin(), residuals.end(),
                    residuals.begin(), residuals.begin(),
                    thrust::multiplies<double>());

  /* Sum up the residuals */
  residual = thrust::reduce(residuals.begin(), residuals.end());

  /* Deallocate memory for residuals vector */
  residuals.clear();

  /* Normalize the residual */
  residual = sqrt(residual / norm);

  return residual;
}


/**
 * @brief Computes the volume-averaged, energy-integrated nu-fission rate in
 *        each FSR and stores them in an array indexed by FSR ID.
 * @details This is a helper method for SWIG to allow users to retrieve
 *          FSR nu-fission rates as a NumPy array. An example of how this method
 *          can be called from Python is as follows:
 *
 * @code
 *          num_FSRs = geometry.getNumFSRs()
 *          fission_rates = solver.computeFSRFissionRates(num_FSRs)
 * @endcode
 *
 * @param fission_rates an array to store the nu-fission rates (implicitly
 *                      passed in as a NumPy array from Python)
 * @param num_FSRs the number of FSRs passed in from Python
 */
void GPUSolver::computeFSRFissionRates(double* fission_rates, int num_FSRs) {

  log_printf(INFO, "Computing FSR fission rates...");

  /* Allocate memory for the FSR nu-fission rates on the device and host */
  FP_PRECISION* dev_fission_rates;
  hipMalloc((void**)&dev_fission_rates, _num_FSRs * sizeof(FP_PRECISION));
  FP_PRECISION* host_fission_rates = new FP_PRECISION[_num_FSRs];

  FP_PRECISION* scalar_flux =
       thrust::raw_pointer_cast(&_scalar_flux[0]);

  /* Compute the FSR nu-fission rates on the device */
  computeFSRFissionRatesOnDevice<<<_B, _T>>>(_FSR_volumes, _FSR_materials,
                                             _materials, scalar_flux,
                                             dev_fission_rates);

  /* Copy the nu-fission rate array from the device to the host */
  hipMemcpy((void*)host_fission_rates, (void*)dev_fission_rates,
             _num_FSRs * sizeof(FP_PRECISION), hipMemcpyDeviceToHost);

  /* Populate the double precision NumPy array for the output */
  for (int i=0; i < _num_FSRs; i++)
    fission_rates[i] = host_fission_rates[i];

  /* Deallocate the memory assigned to store the fission rates on the device */
  hipFree(dev_fission_rates);
  delete [] host_fission_rates;
}
